#include <stdio.h>
#include <hip/hip_runtime.h>
 
__global__ void cuda_sum_kernel(int *a, int *b, int *c, size_t size, float *pos)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    c[idx] = a[idx] + b[idx];
    pos[idx] = pos[idx] / 2;
} 

extern "C" {
void cuda_sum(int *a, int *b, int *c, size_t size, float *pos)
{
    int *d_a, *d_b, *d_c;    
    float *d_pos;

    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));
    hipMalloc((void **)&d_pos, size * sizeof(float));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_pos, pos, size * sizeof(float), hipMemcpyHostToDevice);

    cuda_sum_kernel <<< ceil(size / 256.0), 256 >>> (d_a, d_b, d_c, size, d_pos);

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(pos, d_pos, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_pos);
}
}
