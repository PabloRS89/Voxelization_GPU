
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define MAX_THREADS_BLOCK 1024
 
__global__ void cuda_sum_kernel(size_t size, float *pos)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) {
        return;
    }

    pos[idx] = pos[idx] / 2;
} 

extern "C" {
void cuda_sum(size_t size, float *pos)
{    
    float *d_pos;
    clock_t t_ini,t_fin;

    dim3 BLOCK(ceil(size/MAX_THREADS_BLOCK));
    dim3 THREAD(MAX_THREADS_BLOCK);

    hipMalloc((void **)&d_pos, size * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMemcpy(d_pos, pos, size * sizeof(float), hipMemcpyHostToDevice);

    t_ini=clock();
    hipEventRecord(start);
    
    cuda_sum_kernel <<< BLOCK, MAX_THREADS_BLOCK >>> (size, d_pos);
    //cuda_sum_kernel <<< 2, 1024 >>> (size, d_pos);

    hipEventRecord(stop);
        
    
    hipDeviceSynchronize();
    t_fin=clock();
    printf("%f\n",(double)(t_fin-t_ini)/CLOCKS_PER_SEC);

    hipMemcpy(pos, d_pos, size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_pos);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n",double(milliseconds));
    
}
}
